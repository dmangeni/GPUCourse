#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <stdlib.h>
#include<Windows.h>
#include "../HighPerformanceTimer/HighPerformanceTimer.h"
#include <omp.h>

typedef int array_type_t;
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
void setCuda_device() {
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	if ((cudaStatus = hipSetDevice(0)) != hipSuccess) {
		throw("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
}
void cuda_add(const array_type_t*a, const array_type_t*b, array_type_t*c, unsigned int arraySize) {
	
	//Set the device pointers
	array_type_t* dev_a = 0;
	array_type_t* dev_b = 0;
	array_type_t* dev_c = 0;
	hipError_t cudaStatus;

	//Allocate device memory
	if((cudaStatus = hipMalloc(&dev_a,(sizeof(array_type_t)*arraySize))) != hipSuccess) {
		throw("hipMalloc failed!");
	}

	if ((cudaStatus = hipMalloc(&dev_b, (sizeof(array_type_t)*arraySize))) != hipSuccess) {
		throw("hipMalloc failed!");
	}

	//Copy memory from the host memory to the device
	if ((cudaStatus = hipMemcpy(dev_a,a, (sizeof(array_type_t)*arraySize), hipMemcpyHostToDevice)) != hipSuccess) {
		throw("hipMalloc of dev_a failed!");
	}

	if ((cudaStatus = hipMemcpy(dev_b,b,(sizeof(array_type_t)*arraySize), hipMemcpyHostToDevice)) != hipSuccess) {
		throw("hipMalloc of dev_b failed!");
	}

	//Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, arraySize>>>(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	if ((cudaStatus = hipGetLastError()) != hipSuccess) {
		throw("Add kernel launch failed.");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	if ((cudaStatus= hipDeviceSynchronize()) != hipSuccess) {
		throw("hipDeviceSynchronize returned error after launching addKernel!",);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, arraySize * sizeof(arraySize), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

}
int main()
{
	const unsigned int arraySize = 5;
    const array_type_t a[arraySize] = { 1, 2, 3, 4, 5 };
    const array_type_t b[arraySize] = { 10, 20, 30, 40, 50 };
    array_type_t c[arraySize] = { 0 };

	

	try {
		setCuda_device();
		cuda_add(a, b, c, arraySize);
	}
	catch (char* err_message){
		std::cout << err_message << std::endl;
	}

    /*/ Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

#if	defined(WIN32) || defined(_WIN64)
	system("pause");
#endif
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

   

    /*/ Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }*/

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
