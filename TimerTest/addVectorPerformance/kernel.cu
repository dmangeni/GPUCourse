
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <stdlib.h>
#include<Windows.h>
#include "../HighPerformanceTimer/HighPerformanceTimer.h"
#include <omp.h>


typedef int array_type_t;

bool allocMemory(array_type_t** a, array_type_t**b, int**c, int size) {
	bool retval = false;

	if (!((*a = (array_type_t*)(malloc(sizeof(array_type_t)*size))) == nullptr)) {
		retval = true;
	}
	if (!((*b = (array_type_t*)(malloc(sizeof(array_type_t)*size))) == nullptr)){
		retval = true;
	}
	if (!((*c = (array_type_t*)(malloc(sizeof(array_type_t)*size))) == nullptr)) {
		retval = true;
	}
	
	return retval;
}
void clean_up(array_type_t **a, array_type_t**b, array_type_t**c){
	
	if (!(*a == nullptr))
		free(*a);
	if (!(*b == nullptr))
		free(*b);
	if (!(*c == nullptr))
		free(*c);
}
bool fill_array(array_type_t *a, array_type_t*b, array_type_t*c, int size) {
	for (int i = 0; i < size; i++) {
		array_type_t random_number = (rand() % 100) + 1;
		a[i] = random_number;
		b[i] = random_number;
		c[i] = 0;
	}
	return (!(a == nullptr)||(b == nullptr));
}

void print_arrays(array_type_t *my_array, int size) {
	for (int i = 0; i < size; i++) {
		std::cout << my_array[i] << " ";
		if (i % 5 == 0 && i != 0) {
			std::cout << "\n";
		}
	}
	std::cout<<std::endl;
}
bool addVecSerialCPU(array_type_t *a, array_type_t*b, array_type_t*c, int size) {
	for (int i = 0; i < size; i++) {
		c[i] = a[i] + b[i];
	}
	return (!(c == nullptr));
}

int main(int argc, char*argv[]) {

	//Start timing.
	HighPrecisionTime h;

	int size = 100;
	if (argc > 1) {
		size = std::stoi(argv[1]);
		std::cout << "Size of array: " << size << std::endl;
	
	}
	else {
		std::cout << "ERROR: Usage: nameofprogram sizeofarray" << std::endl;
	}
		
	//Malloc 3 arrays
	array_type_t *a = nullptr;
	array_type_t *b = nullptr;
	array_type_t *c = nullptr;

	try {
		bool malloc_retval = allocMemory(&a, &b, &c, size);
		if (!malloc_retval)
			throw "ERROR: allocating memory for the arrays.";

		//Initialize randomness
		srand(GetTickCount());
		//A program that uses a commandline argument to fill an array
		std::cout << "Filling Arrays with random numbers:" << std::endl;
		if (!fill_array(a, b, c, size)) {
			throw "ERROR: filling arrays with random numbers.";
		}
		
		//startTimer
		const int AVERAGE_TIMES = 100;
		double function_performance = 0;
		h.TimeSinceLastCall();
		for (int i = 0; i < AVERAGE_TIMES; i++) {
			//h.TimeSinceLastCall();
			addVecSerialCPU(a, b, c, size);
			function_performance += h.TimeSinceLastCall();
		}
		function_performance = function_performance / AVERAGE_TIMES;

		std::cout <<"Runtime of addingVectors:: " << std::fixed << function_performance << std::endl;

		/*print_arrays(a, size);
		print_arrays(b, size);*/

		//Add the two vectors
		
	}
	catch (char* err_message) {
		std::cout << err_message << std::endl;
	}

	clean_up(&a, &b, &c);
	system("pause");
	return 0;
}


